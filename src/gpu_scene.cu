#include "hip/hip_runtime.h"
#include "gpu_scene.cuh"
#include <ssaa.cuh>
#include <limits>

namespace render
{

__global__ void PrepareBuffer(Vector3f* deviceData, const int size) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int offset = gridDim.x * blockDim.x;
    for (int i = idx; i < size; i += offset) {
        deviceData[i] = Vector3f(0.0, 0.0, 0.0);
    }
}

void GpuScene::GenerateScene() 
{
    auto texture = Texture(floor_m.TexturePath());
    texture.LoadTextureToDevice();
    auto floor1 = Triangle(floor_m.p1(), floor_m.p3(), floor_m.p2());
    auto floor2 = Triangle(floor_m.p3(), floor_m.p1(), floor_m.p4());

    polygons_m.push_back(
        Polygon(
            floor1, 
            floor_m.color(), 
            floor_m.reflection(), 
            0.0,
            floor_m.p2() - floor_m.p3(),
            floor_m.p2() - floor_m.p1(),
            floor_m.p1() + floor_m.p3() - floor_m.p2(),
            texture));
    polygons_m.push_back(
        Polygon(
            floor2, 
            floor_m.color(), 
            floor_m.reflection(), 
            0.0,
            floor_m.p1() - floor_m.p4(),
            floor_m.p3() - floor_m.p4(),
            floor_m.p4(),
            texture));

    for(uint64_t i = 0; i < OBJECTS_COUNT; ++i)
    {
        objects_m[i].CreateObjectFromFile(ObjectsPaths[i], polygons_m);
    }
    hipMalloc(&devicePolygons_m, polygons_m.size() * sizeof(Polygon));
    hipMemcpy(devicePolygons_m, polygons_m.data(), polygons_m.size() * sizeof(Polygon), hipMemcpyHostToDevice);
}

__global__ void CreateRaysKernel(const Camera* camera, Ray* deviceRays)
{
    double dw = 2.0 / (camera->w() - 1.0);
    double dh = 2.0 / (camera->h() - 1.0);
    double z = 1.0 / std::tan(camera->angle() * M_PI / 360.0);

    Vector3d bz = camera->view() - camera->position();
    Vector3d bx = bz|Vector3d(0.0, 0.0, 1.0);
    Vector3d by = bx|bz;

    bx.norm();
    by.norm();
    bz.norm();

    const int offsetx = blockDim.x * gridDim.x;
    const int offsety = blockDim.y * gridDim.y;

    for (int idx = blockDim.x * blockIdx.x + threadIdx.x; idx < camera->w(); idx += offsetx)
    {
        for (int idy = blockDim.y * blockIdx.y + threadIdx.y; idy < camera->h(); idy += offsety)
        {
            Vector3d v(-1.0 + dw * idx, (-1.0 + dh * idy) * camera->h() / camera->w(), z);
            auto dir = Vector3d::transposeMultiplication(bx, by, bz, v);
            uint64_t rayId = (camera->h() - 1 - idy) * camera->w() + idx;
            deviceRays[idx * camera->h() + idy] = Ray(camera->position(), dir, rayId);
        }
    }    
}

__global__ void ProcessRaysKernel(
    const Ray* deviceRaysInput,
    const int inputSize,
    Ray* deviceRaysOutput,
    int* outputSize,
    Vector3f* deviceData,
    const Light* deviceLights,
    const int lightsCount,
    const Polygon* devicePolygons,
    int polygonsCount)
{
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int offset = gridDim.x * blockDim.x;
    for(int k = idx; k < inputSize; k += offset)
    {
        int minPolygonIndex = -1;
        double minT = 1e18;
        for(int i = 0; i < polygonsCount; ++i)
        {
            bool flag{false};
            auto dpi = devicePolygons[i];
            auto t = dpi.MollerTrumboreIntersection(deviceRaysInput[k], flag);
            if(flag && t < minT)
            {
                minPolygonIndex = i;
                minT = t;
            }
        }
        if(minPolygonIndex < 0)
        {
            continue;
        }
        auto intersection = deviceRaysInput[k].position() + minT * deviceRaysInput[k].view();
        auto dpm = devicePolygons[minPolygonIndex];
        auto color = dpm.ComputeColorInPoint(deviceRaysInput[k], intersection);
        auto phongColor = Phong(
            deviceRaysInput[k],
            intersection,
            minPolygonIndex,
            deviceLights,
            lightsCount,
            devicePolygons,
            polygonsCount
        );
        deviceData[deviceRaysInput[k].uid()] &= phongColor;

        if(dpm.transparency() > 0)
        {
            deviceRaysOutput[atomicAdd(outputSize, 1)] = Ray(
                intersection + SHIFT * deviceRaysInput[k].view(),
                deviceRaysInput[k].view(), 
                deviceRaysInput[k].uid(),
                dpm.transparency() * deviceRaysInput[k].color() * color);
        }

        if(dpm.reflection() > 0)
        {
            auto reflected = Vector3d::reflect(deviceRaysInput[k].view(), dpm.triangle().n());
            deviceRaysOutput[atomicAdd(outputSize, 1)] = Ray(
                intersection + SHIFT * reflected,
                reflected, 
                deviceRaysInput[k].uid(),
                dpm.reflection() * deviceRaysInput[k].color() * color);
        }
    }
}

__global__ void NormalizeDataKernel(Vector3f* deviceData, uchar4* deviceSSAA, int size)
{
    const int offset = gridDim.x * blockDim.x;
    for(int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < size; idx += offset)
    {
        deviceData[idx].range(0.0, 1.0);
        deviceData[idx] *= 255.0;
        deviceSSAA[idx] = make_uchar4(deviceData[idx].x(), deviceData[idx].y(), deviceData[idx].z(), 255);
    }
}


void GpuScene::GenerateFrame(int UUID, uchar4* deviceSSAA)
{
    int w = camera_m.wSSAA();
    int h = camera_m.hSSAA();
    int inputSize = w * h;
    Vector3f* deviceDataPtr;
    hipMalloc(&deviceDataPtr, inputSize * sizeof(Vector3f));
    PrepareBuffer<<<256, 256>>>(deviceDataPtr, inputSize);
    Ray* deviceRaysInputPtr;
    hipMalloc(&deviceRaysInputPtr, inputSize * sizeof(Ray));
    Camera* deviceCameraPtr;
    hipMalloc(&deviceCameraPtr, sizeof(Camera));
    hipMemcpy(deviceCameraPtr, &camera_m, sizeof(Camera), hipMemcpyHostToDevice);

    CreateRaysKernel<<<BLOCKS_2D, THREADS_2D>>>(deviceCameraPtr, deviceRaysInputPtr);
    hipDeviceSynchronize();
    int64_t countRays{0};
    for(int depth = 0; depth < maxDepth; ++depth)
    {
        countRays += inputSize;
        Ray* deviceRaysOutputPtr;
        hipMalloc(&deviceRaysOutputPtr, 2 * inputSize * sizeof(Ray));
        
        int zeroValue = 0;
        int* deviceOutputSizePtr;
        hipMalloc(&deviceOutputSizePtr, sizeof(int));
        hipMemcpy(deviceOutputSizePtr, &zeroValue, sizeof(int), hipMemcpyHostToDevice);

        ProcessRaysKernel<<<256, 256>>>(
            deviceRaysInputPtr, 
            inputSize,
            deviceRaysOutputPtr,
            deviceOutputSizePtr,
            deviceDataPtr,
            deviceLights_m,
            lights_m.size(),
            devicePolygons_m,
            polygons_m.size());
        hipDeviceSynchronize();
        hipMemcpy(&inputSize, deviceOutputSizePtr, sizeof(int), hipMemcpyDeviceToHost);
        hipFree(deviceRaysInputPtr);
        hipFree(deviceOutputSizePtr);
        deviceRaysInputPtr = deviceRaysOutputPtr;
    }
    NormalizeDataKernel<<<256, 256>>>(deviceDataPtr, deviceSSAA, w * h);
    hipDeviceSynchronize();
    hipFree(deviceDataPtr);
    hipFree(deviceRaysInputPtr);
    hipFree(deviceCameraPtr);
} 

void GpuScene::Render() 
{
    double dt = 2 * M_PI / frames_m;
    camera_m.setSSAA(upscaleFactor);
    uchar4* deviceSSAA;
    hipMalloc(&deviceSSAA, camera_m.wSSAA() * camera_m.hSSAA() * sizeof(uchar4));
    uchar4* deviceData;
    hipMalloc(&deviceData, camera_m.w() * camera_m.h() * sizeof(uchar4));
    for(int k = 0; k < frames_m; ++k)
    {
        double time = k * dt;
        camera_m.updatePosition(time);
        camera_m.updateView(time);
        GenerateFrame(k, deviceSSAA);
        ssaaGPUKernel<<<BLOCKS_2D, THREADS_2D>>>(
            deviceSSAA, 
            deviceData,
            camera_m.w(),
            camera_m.h(),
            upscaleFactor);
        std::vector<uchar4> output(camera_m.w() * camera_m.h());
        hipMemcpy(output.data(), deviceData, camera_m.w() * camera_m.h() * sizeof(uchar4), hipMemcpyDeviceToHost);
        SaveFile(savePath_m, output, camera_m.w(), camera_m.h(), k);
    }

    hipFree(deviceSSAA);
    hipFree(deviceData);
}

}